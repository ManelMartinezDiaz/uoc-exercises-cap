
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <string.h>


void initData(void);
void initValues(void);
void initValuesLin(void);

void calculateMovingAverage(void);
void calculateMovingAverageCuda(void);
__global__ void kernelMovingAverage(double *MOV_LIN_D, double *DATA_LIN_D, int *ROWS);

void calculateColumnAverage(void);
void calculateColumnAverageCuda(void);
__global__ void kernelColumnAverage(double *AVG_D, double *DATA_LIN_D, int *COLUMNS, int *ELEMS_AVG);

void showData(void);
void showMov(void);
void showColAverage(void);
void showMovCuda(void);
void showColAverageCuda(void);

void createFilesData(void);
void createFilesMov(void);
void createFilesAvg(void);


void checkResults(void);

void freeAll(void);

double **DATA;
double **MOV;

double *DATA_LIN;
double *DATA_TRANSPOSED_LIN;
double *MOV_LIN;

double *DATA_TRANSPOSED_LIN_D;
double *AVG_D;

double *AVG;
double *AVG_CUDA;

const int OK = 1;
const int KO = 0;

int COLUMNS;
int ROWS;
int ELEMS_AVG;
int TOTAL_CELLS;

void initData(void) {
	int i;

	AVG = (double *) malloc(COLUMNS * sizeof(double));
	AVG_CUDA = (double *) malloc(COLUMNS * sizeof(double));
	DATA = (double **) malloc(ROWS * sizeof(double *));
	MOV = (double **) malloc(ROWS * sizeof(double *));
	DATA_LIN  = (double *) malloc(TOTAL_CELLS * sizeof(double));
	DATA_TRANSPOSED_LIN = (double *) malloc(TOTAL_CELLS * sizeof(double));
	MOV_LIN = (double *) malloc(TOTAL_CELLS * sizeof(double));

	for (i = 0; i < ROWS; i++) {
		DATA[i] = (double *) malloc(COLUMNS * sizeof(double));
		MOV[i] = (double *) malloc(COLUMNS * sizeof(double *));
	}

	initValues();
}

void initValues(void) {
	int i, j;

	srand(time(NULL));

	for (i = 0; i < ROWS; i++) {
		for (j = 0; j < COLUMNS; j++) {
			DATA[i][j] = (100.0 * rand()) / ((double) RAND_MAX);
			MOV[i][j] = 0.0;
		}
	}

	for (i = 0; i < COLUMNS; i++) {
		AVG[i] = 0.0;
		AVG_CUDA[i] = 0.0;
	}

	initValuesLin();
}

void initValuesLin(void) {
	int i, j, k, l;

	srand(time(NULL));

	for (i = 0; i < ROWS; i++) {
		for (j = 0; j < COLUMNS; j++) {
			k = i * COLUMNS + j;
			l = j * ROWS + i;
			DATA_LIN[k] = DATA[i][j];
			MOV_LIN[k] = MOV[i][j];
			DATA_TRANSPOSED_LIN[l] = DATA[i][j];
		}
	}
}

int canShow() {
	if ((ROWS <= 15) && (COLUMNS <= 15)) {
		return 1;
	} else {
		return 0;
	}
}

void showData(void) {
	int x, y;

	if (!canShow()) return;

	for (y = 0; y < ROWS; y++) {
		for (x = 0; x < COLUMNS; x++) {
			printf("%07.4f ", DATA[y][x]);
		}
		printf("\n");
	}
	printf("\n");
}

void showMov(void) {
	int x, y;

	if (!canShow()) return;

	for (y = 0; y < ROWS; y++) {
		for (x = 0; x < COLUMNS; x++) {
			printf("%07.4f ", MOV[y][x]);
		}
		printf("\n");
	}
	printf("\n");
}

void showMovCuda(void)  {
	int x, y;

	if (!canShow()) return;

	for (y = 0; y < ROWS; y++) {
		for (x = 0; x < COLUMNS; x++) {
			printf("%07.4f ", MOV_LIN[y * COLUMNS + x]);
		}
		printf("\n");
	}
	printf("\n");

}

void showColAverage(void) {
	int x;

	if (!canShow()) return;

	for (x = 0; x < COLUMNS; x++) {
		printf("%07.4f ", AVG[x]);
	}
	printf("\n\n");
}

void showColAverageCuda(void) {
	int x;

	if (!canShow()) return;

	for (x = 0; x < COLUMNS; x++) {
		printf("%07.4f ", AVG_CUDA[x]);
	}
	printf("\n\n");
}

void calculateMovingAverage(void) {
        int x, y;
	int i;
	int elems; 

	for (y = 0; y < ROWS; y++) {
		for (x = 0; x < COLUMNS; x++) {
	                MOV[y][x] = DATA[y][x];
		        elems = 1;
	                for(i = 1; i < ELEMS_AVG; i++) {
	                        if ((x - i) >= 0) {
		                        MOV[y][x] = MOV[y][x] + DATA[y][x - i];
			                elems++;
		                }
	                }
                        MOV[y][x] = MOV[y][x] / elems;
		}
	}
}

void calculateColumnAverage(void) {
    int x, y;

	for (x = 0; x < COLUMNS; x++) {
		for (y = 0; y < ROWS; y++) {
			AVG[x] = AVG[x] + DATA[y][x];
		}
	    AVG[x] = AVG[x] / ROWS;
	}
}

__global__ void kernelMovingAverage(double *MOV_LIN_D, double *DATA_LIN_D, int *COLUMNS, int *ELEMS_AVG) {
	int i;
	int posIni;
	double cell;
        int columns;
        int elemsAvg;
	int elems;
        int row;
        int column;

        columns = *COLUMNS;
        elemsAvg = *ELEMS_AVG;
        row = blockIdx.x;
        column = threadIdx.x;

	cell = 0.0;
        elems = 1;

	posIni = row * columns + column;
	cell = DATA_LIN_D[posIni];

        for(i = 1; i < elemsAvg; i++) {
            if ((column - i) >= 0) {
                cell = cell + DATA_LIN_D[posIni - i];
		elems++;
            }
        }
	
        cell = cell / elems;

	MOV_LIN_D[posIni] = cell;
}

void calculateMovingAverageCuda() {
        double *DATA_LIN_D;
        double *MOV_LIN_D;
	int *COLUMNS_D;
        int *ELEMS_AVG_D;

        hipMalloc((void **) &DATA_LIN_D, TOTAL_CELLS * sizeof(double));
	hipMalloc((void **) &MOV_LIN_D, TOTAL_CELLS * sizeof(double));
        hipMalloc((void **) &COLUMNS_D, sizeof(int));
        hipMalloc((void **) &ELEMS_AVG_D, sizeof(int));

	hipMemcpy(DATA_LIN_D, DATA_LIN, TOTAL_CELLS * sizeof(double), hipMemcpyHostToDevice );
        hipMemcpy(MOV_LIN_D, MOV_LIN, TOTAL_CELLS * sizeof(double), hipMemcpyHostToDevice );
        hipMemcpy(COLUMNS_D, &COLUMNS, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(ELEMS_AVG_D, &ELEMS_AVG, sizeof(int), hipMemcpyHostToDevice);

        dim3 numColumns(COLUMNS, 1);
        dim3 numRows(ROWS, 1);
        kernelMovingAverage<<<numRows, numColumns>>>(MOV_LIN_D, DATA_LIN_D, COLUMNS_D, ELEMS_AVG_D);

	hipMemcpy(MOV_LIN, MOV_LIN_D, TOTAL_CELLS * sizeof(double), hipMemcpyDeviceToHost);

        hipFree(COLUMNS_D);
        hipFree(ELEMS_AVG_D);
	hipFree(MOV_LIN_D);
	hipFree(DATA_LIN_D);
}

__global__ void kernelColumnAverage(double *AVG_D, double *DATA_LIN_D, int *ROWS) {
	int i;
	int posIni;
	int posFin;
        int rows;
	double cell;

        rows = *ROWS;

	cell = 0.0;

	posIni = threadIdx.x * rows;
	posFin = posIni + rows;

	for (i = posIni; i < posFin; i++) {
		cell = cell + DATA_LIN_D[i];
	}

	cell = cell / rows;

	AVG_D[threadIdx.x] = cell;
}

void calculateColumnAverageCuda(void) {
        double *DATA_TRANSPOSED_LIN_D;
        double *AVG_D;
        int *ROWS_D;

	hipMalloc((void **) &DATA_TRANSPOSED_LIN_D, TOTAL_CELLS * sizeof(double));
	hipMalloc((void **) &AVG_D, COLUMNS * sizeof(double));
        hipMalloc((void **) &ROWS_D, sizeof(int));

	hipMemcpy(DATA_TRANSPOSED_LIN_D, DATA_TRANSPOSED_LIN, TOTAL_CELLS * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(AVG_D, AVG, COLUMNS * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(ROWS_D, &ROWS, sizeof(int), hipMemcpyHostToDevice);

        dim3 numColumns(COLUMNS, 1);
        dim3 numRows(1, 1);
	kernelColumnAverage<<<numRows, numColumns>>>(AVG_D, DATA_TRANSPOSED_LIN_D, ROWS_D);

	hipMemcpy(AVG_CUDA, AVG_D, COLUMNS * sizeof(double), hipMemcpyDeviceToHost);

        hipFree(ROWS_D);
	hipFree(DATA_TRANSPOSED_LIN_D);
	hipFree(AVG_D);
}

void  checkResults(void) {
	int y,x, k;
	double diff;
	double epsilon;
	int result;

	result= OK;
	epsilon = 0.0000001;

	for(y = 0; y < ROWS; y++) {
		for(x = 0; x < COLUMNS; x++) {
			k = y * COLUMNS + x;
			diff = abs(MOV[y][x] - MOV_LIN[k]);

			if (diff > epsilon) {
				result = KO;
				break;
			}
		}
	}

	printf("\nCheck result moving average : %d\n\n", result);

	result= OK;

	for(x = 0; x < COLUMNS; x++) {
	    diff = abs(AVG[x] - AVG_CUDA[x]);

	    if (diff > epsilon) {
		    result = KO;
			break;
		}
	}

	printf("Check result column average : %d\n\n", result);
}


void freeAll(void) {
	int y;

	free(AVG);
	for (y = 0; y < ROWS; y++) {
		free(DATA[y]);
		free(MOV[y]);
	}

	free(DATA);
	free(MOV);
}

void createFilesData(void) {
	int i, j, k;
	char line[1024];
	char fileName[] = "DATA_0.txt";

	k = 0;
	FILE *fd;
	for (i = 0; i < ROWS; i++) {
		fileName[5] = '1' + i;
		fd = fopen(fileName, "w");
		for (j = 0; j < COLUMNS; j++) {
			fgets(line, sizeof(line), fd );
			k = i * COLUMNS + j;
			sprintf(line, "%f\n", DATA_LIN[k]);
			fputs(line, fd);
		}
		fclose(fd);
	}
}

void createFilesMov(void)  {
	int i, j, k;
	char line[1024];
	char fileName[] = "MOV_0.txt";

	k = 0;
	FILE *fd;
	for (i = 0; i < ROWS; i++) {
		fileName[4] = '1' + i;
		fd = fopen(fileName, "w");
		for (j = 0; j < COLUMNS; j++) {
			k = i * COLUMNS + j;
			sprintf(line, "%f\n", MOV_LIN[k]);
			fputs(line, fd);
		}
		fclose(fd);
	}
}

void createFilesAvg(void)  {
	int i;
	char line[1024];
	char fileName[] = "AVG.txt";

	FILE *fd;

	fd = fopen(fileName, "w");

	for (i = 0; i < COLUMNS; i++) {
		sprintf(line, "%f\n", AVG_CUDA[i]);
		fputs(line, fd);
	}

        fclose(fd);	
}

int main(int argc, char**argv) {
	if (argc != 4) {
		printf("usage: exercici_cuda elems_avg N M\n elems_avg : number of elements to consider in moving average calculations\n N : number of rows\n M number of columns\n\n");
		ELEMS_AVG = 9;
	        ROWS = 5;
	        COLUMNS = 10;
	} else {
		ELEMS_AVG = atoi(argv[1]);
	        ROWS = atoi(argv[2]);
	        COLUMNS = atoi(argv[3]);
	}

	TOTAL_CELLS = ROWS * COLUMNS;

	printf("begin\n");

	initData();

	showData();

	createFilesData();
	
	calculateMovingAverage();
	showMov();

	calculateColumnAverage();
	showColAverage();

	calculateMovingAverageCuda();
	showMovCuda();

	createFilesMov();

	calculateColumnAverageCuda();
	showColAverageCuda();

	createFilesAvg();
	
	checkResults();

	freeAll();

	printf("end\n");	
}
