
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 1000 // Nº Columnes
#define M 10  // Nº Files
#define ELE 4 // Elements anteriors

__global__ void moving_average(float *a, float *b) {

	int index = blockDim.x * blockIdx.x + threadIdx.x;
	float result;

	if (index % N >= ELE){ //El primer valor de cada fila a calcular és el Nº ELE
		for(int i=0;i < ELE;i++) {
		result = result + a[index-i];
		}
		b[index] = result/ELE;
	} else {
		b[index] = a[index];
	}
}

__global__ void average_col(float *a, float *b) {

	int index = blockDim.x * blockIdx.x + threadIdx.x;
	int column = threadIdx.x;  //  Tenim un thread per columna segons definit a DimBlock (ThreadsPerBlock)
	float result;

	// Recorrer els valors en vertical de la matriu DADES i fer mitjana
	// Per a cada columna (column = nº threads per simplificar) es suma el valor de totes les files (N*i)
	for(int i=0; i<M; i++){
		result = result + a[N*i+column];
	}

        b[index] = result/M;  // Es fa la mitjana amb el nº de files
	
}


int main()
{
	// Definim els punters de les matrius/vector (DADES, MOV i AVG) al host i device
	float *host_data, *host_mov, *host_avg;
	float *device_data, *device_mov, *device_avg;

	const int size_host_data = N*M*sizeof(float);
	const int size_host_mov = N*M*sizeof(float);
	const int size_host_avg = N*sizeof(float);

	// Assignem memòria per a les matrius/vector al host i device
	host_data = (float*)malloc(size_host_data);
	host_mov = (float*)malloc(size_host_mov);
	host_avg = (float*)malloc(size_host_avg);

	hipMalloc((void**)&device_data, size_host_data);
	hipMalloc((void**)&device_mov, size_host_mov);
	hipMalloc((void**)&device_avg, size_host_avg);

        // Inicilitzem les matrius
	// Matriu DATA[N,M] --> host_data --> Valors aletaris del 0 al 10
	// Matriu MOV[N,M] --> host_mov --> '0'
	// Vector AVG[N] --> host_avg --> '0'
	int i, j;

	for(i=0;i<N*M;i++){ host_data[i]= (float)(rand() % 10);} 
        for(i=0;i<N*M;i++){ host_mov[i]= 0;}
        for(j=0;j<N;j++){ host_avg[j]= 0;}

	//
	printf("Resultat execució matriu DADES[%i,%i].\n", N, M);

        // Mostrar matriu DADES
        printf("Matriu DADES[N,M]:\n");
        for(int i=0;i<M;i++){
                printf("Row Nº%i |",i);
                for(int j=0;j<N;j++){
                        printf(" %f |",host_data[i*N+j]);
                }
                printf("\n");
        }

	// Copiem les matrius/vector del host al device
	hipMemcpy( device_data, host_data, size_host_data, hipMemcpyHostToDevice);
	hipMemcpy( device_mov, host_mov, size_host_mov, hipMemcpyHostToDevice);
	hipMemcpy( device_avg, host_avg, size_host_avg, hipMemcpyHostToDevice);
	
	// Definim el nº de blocs, el nº de fluxos i s'invoca el kernel per calcular el 'Moving Average'
	// Creem un únic bloc i un thread per a cada 
        int blocksPerGrid = 1; // Nº de blocs
	int threadsPerBlock = N * M; // Nº de blocs
	moving_average<<<blocksPerGrid, threadsPerBlock>>>(device_data, device_mov);

	hipMemcpy( host_mov, device_mov, size_host_mov, hipMemcpyDeviceToHost );
	hipFree( host_mov);

        // Definim el nº de blocs, el nº de fluxos i s'invoca el kernel per calcular el	'Average Column'
	// La matriu dades encara està al dispositiu
	// Creem un únic bloc i un thread per cada columna
        blocksPerGrid = 1;
        threadsPerBlock = N;
	average_col<<<blocksPerGrid, threadsPerBlock>>>(device_data,device_avg);
        hipMemcpy( host_avg, device_avg, size_host_avg, hipMemcpyDeviceToHost );
	hipFree( host_data);
	hipFree( host_avg);

        // Mostrar matriu MOV
        printf("Matriu MOV[N,M]:\n");
        for(int i=0;i<M;i++){
                printf("Row Nº%i |",i);
                for(int	j=0;j<N;j++){
                        printf(" %f |",host_mov[i*N+j]);
                }
                printf("\n");
        }

        // Mostrar vector AVG
        printf("Matriu AVG[N]:\n");
	printf("Average: ");
        for(int i=0;i<N;i++){
        	printf(" %f |",host_avg[i]);
        }
        printf("\n");

return EXIT_SUCCESS; 
}
